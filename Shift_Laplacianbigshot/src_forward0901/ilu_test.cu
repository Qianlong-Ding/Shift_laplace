#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <common.h>

#include "time.h"
#include "par.h"
#include "su.h"
#include "segy.h"
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include <mpi.h>
#include <assert.h>
#include <ctype.h>
#include "hipsolver.h"

#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"

int main(int argc, char **argv)
{
    int ix;
    int iz;
    int nx;
    int nz;
    int no_zero;


    hipFloatComplex *h_A;
    hipFloatComplex *h_CsrValA;
    hipFloatComplex *h_b;
    hipFloatComplex *h_x;

    hipFloatComplex *d_A;
    hipFloatComplex *d_csrVal;
    hipFloatComplex *d_b;
    hipFloatComplex *d_x;

    int *h_CsrColIndA;
    int *h_CsrRowPtrA;

    int *d_csrColInd;
    int *d_csrRowPtr;

    hipFloatComplex alpha;
    hipFloatComplex beta;

    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    hipsparseMatDescr_t descr_U = 0;
    csrilu02Info_t info_M;
    csrsv2Info_t  info_L;
    csrsv2Info_t info_U;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_U;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    //const double alpha = 1.;
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseStatus_t status;
    int m,nnz;

    hipFloatComplex *d_z;
    hipFloatComplex *d_y;

    hipsparseCreate(&handle);

    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    nx=nz=4;

    hipMalloc((hipFloatComplex **)&d_z,      sizeof(hipFloatComplex)*nx);
    hipMalloc((hipFloatComplex **)&d_y,      sizeof(hipFloatComplex)*nx);


    h_A = (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * (nx)*(nx));
    h_CsrValA= (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * (nx)*(nx));

    h_b= (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * (nx));
    h_x= (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * (nx));

    h_CsrColIndA= (int *)malloc(sizeof(int) * (nx)*(nx));
    h_CsrRowPtrA= (int *)malloc(sizeof(int) * (nx+1));

    memset(h_A     ,         0, sizeof(hipFloatComplex)*(nx)*(nx));
    memset(h_CsrValA,        0, sizeof(hipFloatComplex)*(nx)*(nx));
    memset(h_b     ,         0, sizeof(hipFloatComplex)*(nx));
    memset(h_x     ,         0, sizeof(hipFloatComplex)*(nx));

    memset(h_CsrColIndA ,        0, sizeof(int)*(nx)*(nx));
    memset(h_CsrRowPtrA ,        0, sizeof(int)*(nx+1));

    hipMalloc((hipFloatComplex **)&d_csrVal,      sizeof(hipFloatComplex)*nx*nx);
    hipMalloc((hipFloatComplex **)&d_b,            sizeof(hipFloatComplex)*nx);
    hipMalloc((hipFloatComplex **)&d_x,            sizeof(hipFloatComplex)*nx);
    hipMalloc((int **)&d_csrColInd,              sizeof(int)*nx*(nx));
    hipMalloc((int **)&d_csrRowPtr,              sizeof(int)*(nx+1));


    /*hipMemset(d_CsrValA,            0, sizeof(hipComplex)*nx);
    hipMemset(d_b,            0, sizeof(hipComplex)*nx);
    hipMemset(d_x,            0, sizeof(hipComplex)*nx);
    hipMemset(d_x,            0, sizeof(hipComplex)*nx);
    hipMemset(d_x,            0, sizeof(hipComplex)*nx);*/

    //hipMemset(d_CsrValA,            0, sizeof(hipComplex)*nx*nx);

    ix=0;
    iz=0;
    h_A[ix*nx+iz].x=2.0;h_A[ix*nx+iz].y=2.0; ix=1;iz=0; h_A[ix*nx+iz].x=2.0;h_A[ix*nx+iz].y=2.0;

    ix=0;
    iz=1;
    h_A[ix*nx+iz].x=3.0;h_A[ix*nx+iz].y=3.0; ix=1;iz=1; h_A[ix*nx+iz].x=2.0;h_A[ix*nx+iz].y=2.0; ix=2;iz=1; h_A[ix*nx+iz].x=4.0;h_A[ix*nx+iz].y=4.0;

    ix=1;
    iz=2;
    h_A[ix*nx+iz].x=5.0;h_A[ix*nx+iz].y=5.0; ix=2;iz=2; h_A[ix*nx+iz].x=1.0;h_A[ix*nx+iz].y=1.0; ix=3;iz=2; h_A[ix*nx+iz].x=3.0;h_A[ix*nx+iz].y=3.0;


    ix=2;
    iz=3;
    h_A[ix*nx+iz].x=2.0;h_A[ix*nx+iz].y=2.0; ix=3;iz=3; h_A[ix*nx+iz].x=3.0;h_A[ix*nx+iz].y=3.0;

    no_zero=0;

    for(iz=0;iz<nz;iz++)
    {
        for(ix=0;ix<nx;ix++)
        {
            if((h_A[ix*nx+iz].x*h_A[ix*nx+iz].x)>1e-6)
            {
                h_CsrValA[no_zero].x=h_A[ix*nx+iz].x;
                h_CsrValA[no_zero].y=h_A[ix*nx+iz].y;
                printf("h_CsrValA[no_zero].x=%f\n",h_CsrValA[no_zero].x);
                h_CsrColIndA[no_zero]=ix;
                no_zero++;
            }
            
        }
        h_CsrRowPtrA[iz+1]=no_zero;
        h_b[iz].x=1.0;
        h_x[iz].x=1.0;
    }

    printf("no_zero=%d\n",no_zero);



    hipMemcpy(d_csrVal,h_CsrValA ,sizeof(hipFloatComplex)*nx*nx,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b ,sizeof(hipFloatComplex)*nx,hipMemcpyHostToDevice);
    hipMemcpy(d_x,h_x ,sizeof(hipFloatComplex)*nx,hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd,h_CsrColIndA ,sizeof(int)*nx*(nx),hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtr,h_CsrRowPtrA ,sizeof(int)*(nx+1),hipMemcpyHostToDevice);

    hipMemset(d_b,            0, sizeof(hipComplex)*nx);
    alpha.x=1.0;
    alpha.y=0.0;
    beta.x=0.0;
    beta.y=0.0;
    //y = alpha * op(A) * x  + beta * y
    //HIPSPARSE_OPERATION_TRANSPOSE
    //HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE
    //HIPSPARSE_OPERATION_NON_TRANSPOSE
    hipsparseCcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nz,nx,no_zero,&alpha,
        descr,d_csrVal,d_csrRowPtr,d_csrColInd,
            d_x,&beta,d_b);
    /*hipsparseCcsrmv(handle,HIPSPARSE_OPERATION_TRANSPOSE,nz,nx,no_zero,&alpha,
        descr,d_CsrValA,d_CsrRowPtrA,d_CsrColIndA,
            d_x,&beta,d_b);*/

    (hipMemcpy(h_b,d_b,sizeof(hipFloatComplex)*nx,hipMemcpyDeviceToHost));

    for(ix=0;ix<nx;ix++)
    {
        printf("d_b[%d]=%f+%fi\n",ix,h_b[ix].x,h_b[ix].y);
    }

    m=nx;
    nnz=no_zero;

    checkCudaErrors(hipMemcpy(h_CsrValA,d_csrVal,sizeof(hipFloatComplex)*nx*nx,hipMemcpyDeviceToHost));
for(ix=0;ix<no_zero;ix++)
    {
        //printf("ilu [%d]=%f\n",ix,h_CsrValA[ix].x);
    }

checkCudaErrors(hipsparseCreateMatDescr(&descr_M));
hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

hipsparseCreateMatDescr(&descr_L);
hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

hipsparseCreateMatDescr(&descr_U);
hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

hipsparseCreateCsrilu02Info(&info_M);
hipsparseCreateCsrsv2Info(&info_L);
hipsparseCreateCsrsv2Info(&info_U);

hipsparseCcsrilu02_bufferSize(handle, m, nnz,
    descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M);

hipsparseCcsrsv2_bufferSize(handle, trans_L, m, nnz,
    descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L, &pBufferSize_L);

hipsparseCcsrsv2_bufferSize(handle, trans_U, m, nnz,
    descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U, &pBufferSize_U);

pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

hipMalloc((void**)&pBuffer, pBufferSize);

hipsparseCcsrilu02_analysis(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M,
    policy_M, pBuffer);

status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &structural_zero);
if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
   printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
}

hipsparseCcsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
    d_csrVal, d_csrRowPtr, d_csrColInd,
    info_L, policy_L, pBuffer);
hipsparseCcsrsv2_analysis(handle, trans_U, m, nnz, descr_U,
    d_csrVal, d_csrRowPtr, d_csrColInd,
    info_U, policy_U, pBuffer);

// step 5: M = L * U
hipsparseCcsrilu02(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer);

status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &numerical_zero);
if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
   printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
}


checkCudaErrors(hipMemcpy(h_CsrValA,d_csrVal,sizeof(hipFloatComplex)*nx*nx,hipMemcpyDeviceToHost));

    for(ix=0;ix<no_zero;ix++)
    {
        printf("ilu [%d]=%f+%fi\n",ix,h_CsrValA[ix].x,h_CsrValA[ix].y);
    }


// step 6: solve L*z = x


/*checkCudaErrors(hipsparseCcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
   d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
   d_b, d_z, policy_L, pBuffer));

memset( h_x,        0, sizeof(hipComplex)*(nx));
checkCudaErrors(hipMemcpy(h_x,d_z,sizeof(hipFloatComplex)*nx,hipMemcpyDeviceToHost));

    for(ix=0;ix<nx;ix++) 
    {
        printf("ilu d_z[%d]=%f+%fi\n",ix,h_x[ix].x,h_x[ix].y);
    }
// step 7: solve U*y = z

checkCudaErrors(hipsparseCcsrsv2_solve(handle, trans_U, m, nnz, &alpha, descr_U,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_U,
    d_z, d_y, policy_U, pBuffer));

memset( h_x,        0, sizeof(hipComplex)*(nx));
checkCudaErrors(hipMemcpy(h_x,d_y,sizeof(hipFloatComplex)*nx,hipMemcpyDeviceToHost));

    for(ix=0;ix<nx;ix++)
    {
        printf("ilu d_y[%d]=%f+%fi\n",ix,h_x[ix].x,h_x[ix].y);
    }*/




// step 6: free resources
hipFree(pBuffer);
hipsparseDestroyMatDescr(descr_M);
hipsparseDestroyMatDescr(descr_L);
hipsparseDestroyMatDescr(descr_U);
hipsparseDestroyCsrilu02Info(info_M);
hipsparseDestroyCsrsv2Info(info_L);
hipsparseDestroyCsrsv2Info(info_U);





    free(h_A);
    free(h_CsrValA);
    free(h_b);
    free(h_x);
    free(h_CsrColIndA);
    free(h_CsrRowPtrA);

    hipFree(d_csrVal);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_csrColInd);
    hipFree(d_csrRowPtr);
    hipFree(d_y);
    hipFree(d_z);

    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(handle);





}